#include "hip/hip_runtime.h"
/** Detray tutorial project, No copy right **/

// Project include(s).
#include "propagation_cuda.hpp"

// Detray include(s).
#include "detray/definitions/cuda_definitions.hpp"

__global__ void cuda_propagation_kernel(
    detector_view<detector_host_type> det_data,
    const constant_magnetic_field<> B_field,
    vecmem::data::vector_view<free_track_parameters> tracks_data,
    vecmem::data::jagged_vector_view<intersection_t> candidates_data)
{
    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    detector_device_type det(det_data);
    vecmem::device_vector<free_track_parameters> tracks(tracks_data);
    vecmem::jagged_device_vector<intersection_t> candidates(candidates_data);

    if (gid >= tracks.size())
    {
        return;
    }

    // Create RK stepper
    rk_stepper_type s(B_field);

    // Create navigator
    navigator_device_type n(det);

    // Create propagator
    propagator_device_type propagator(std::move(s), std::move(n));

    // Create the propagator state
    propagator_device_type::state state(
        tracks.at(gid), actor_chain<>::state{}, candidates.at(gid));

    // Run propagation
    propagator.propagate(state);
}

// CUDA propagation function
void cuda_propagation(
    detector_view<detector_host_type> det_data,
    const constant_magnetic_field<> B,
    vecmem::data::vector_view<free_track_parameters> &tracks_data,
    vecmem::data::jagged_vector_view<intersection_t> &candidates_data)
{

    constexpr int thread_dim = 2 * WARP_SIZE;
    int block_dim = tracks_data.size() / thread_dim + 1;

    // run the test kernel
    cuda_propagation_kernel<<<block_dim, thread_dim>>>(
        det_data, B, tracks_data, candidates_data);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}